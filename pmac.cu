#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>
#include "aes.h"


using namespace std;


__global__ void first(uint8_t* d_message, uint8_t* d_key1, int* d_key2, uint8_t* d_cipher){ 

  __shared__ uint8_t cipher[16];

  // parallal part
  uint8_t message_block[16];
  int value = ((*d_key2)*(threadIdx.x))%1000000007;
  for(size_t i = 0; i < 16; ++i){
    message_block[i] = d_message[threadIdx.x*16+i]; 
    message_block[i] ^= ((value >> (i * 8)) & 0xFF);
  }
  uint8_t *cipher_block;

  AES aes(AESKeyLength::AES_128);
  cipher_block = aes.EncryptECB(message_block, 16, d_key1);

  for(size_t i = 0; i < 16; ++i){
    cipher[i] ^= cipher_block[i]; 
  }

  // sync
  __syncthreads();

  for(size_t i = 0; i < 16; ++i){
    d_cipher[i] = cipher[i]; 
  }

}

__global__ void second(uint8_t* d_cipher, uint8_t* d_key1, uint8_t* d_final){ 

  AES aes(AESKeyLength::AES_128);
  

  uint8_t *final_cipher;
  final_cipher = aes.EncryptECB(d_cipher, 16, d_key1);

  for(int i = 0; i < 16; i++){
    d_final[i] = final_cipher[i];
  }

}


int main(int argc, char* argv[])
{
    // reading plaintext files
    ifstream message_file(argv[1], ios::binary);

    message_file.seekg(0, ios::end);
    size_t file_size = message_file.tellg();
    size_t padding_bytes = 16 - (file_size % 16);
    size_t message_size = file_size + padding_bytes;
    message_file.seekg(0, ios::beg);

    uint8_t message[message_size];
    message_file.read(reinterpret_cast<char*>(message), file_size);
    message_file.close();

    message[file_size] = 0x80; 
    for (size_t i = file_size + 1; i < message_size; ++i) {
        message[i] = 0x00; 
    }


    // reading first key file
    ifstream key_file1(argv[2], ios::binary);

    key_file1.seekg(0, ios::end);
    file_size = key_file1.tellg();
    if(file_size != 128){
        cerr << "KEY SHOULD BE OF 16 BYTES ONLY!" << endl;
        return 1;
    }
    key_file1.seekg(0, ios::beg);

    uint8_t key1[16];
    key_file1.read(reinterpret_cast<char*>(key1), 16);
    key_file1.close();


    // reading second key file
    ifstream key_file2(argv[3]);
    int key2;
    key_file2 >> key2;
    key_file2.close();

    // uint8_t key2[16];
    // key_file2.read(reinterpret_cast<char*>(key2), 16);
    // key_file2.close();

    // parallal computation
    uint8_t *d_message, *d_key1, *d_cipher, *d_final;
    hipMalloc(&d_message, message_size*sizeof(uint8_t));
    hipMalloc(&d_key1, 16*sizeof(uint8_t));
    hipMalloc(&d_cipher, 16*sizeof(uint8_t));
    hipMalloc(&d_final, 16*sizeof(uint8_t));

    hipMemcpy(d_message, message, message_size*sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_key1, key1, 16*sizeof(uint8_t), hipMemcpyHostToDevice);

    int *d_key2;
    hipMalloc((void **)&d_key2, sizeof(int));
    hipMemcpy(d_key2, &key2, sizeof(int), hipMemcpyHostToDevice); 

    first<<<1, message_size/16>>>(d_message, d_key1, d_key2, d_cipher);
    hipDeviceSynchronize();


    // second step of pmac
    second<<<1, 1>>>(d_cipher, d_key1, d_final);
    hipDeviceSynchronize();

    uint8_t* h_cipher = (uint8_t*)malloc(16* sizeof(uint8_t));
    hipMemcpy(h_cipher, d_final, 16*sizeof(uint8_t), hipMemcpyDeviceToHost);

    // saving output
    ofstream outFile("tag.txt");
    for (size_t i = 0; i < 16; ++i) {
        outFile << std::hex << static_cast<int>(h_cipher[i]) << " ";
    }
    outFile.close();



    // // thrust

    // // thrust::host_vector<uint8_t> h_message(message, message+message_size);
    // // thrust::host_vector<uint8_t> h_key1(key1, key1+16);
    // // thrust::host_vector<uint8_t> h_key2(key2, key2+16);

    // // thrust::host_vector<uint8_t> h_cipher(16, 0);

    // // thrust::device_vector<uint8_t> d_message(message, message+message_size);
    // // thrust::device_vector<uint8_t> d_key1(key1, key1+16);
    // // thrust::device_vector<uint8_t> d_key2(key2, key2+16);

    // // thrust::device_vector<uint8_t> d_cipher(16, 0);

    // // fun<<<1, message_size/16>>>(d_message, d_key1, d_key2, d_cipher);


    // // fun<<<1, message_size/16>>>(thrust::raw_pointer_cast(d_message.data()),
    // //                                          thrust::raw_pointer_cast(d_key1.data()),
    // //                                          thrust::raw_pointer_cast(d_key2.data()),
    // //                                          thrust::raw_pointer_cast(d_cipher.data()));
    // // hipDeviceSynchronize();


    // // shared

    // // sycl::queue queue(sycl::default_selector{});
    // // uint8_t* d_message = sycl::malloc_shared<int>(file_size, queue);
    // // uint8_t* d_key1 = sycl::malloc_shared<int>(16, queue);
    // // uint8_t* d_key2 = sycl::malloc_shared<int>(16, queue);
    // // uint8_t* d_cipher = sycl::malloc_shared<int>(16, queue);

    // // d_message = message;
    // // d_key1 = key1;
    // // d_key2 = key2;



   

   

    
}
